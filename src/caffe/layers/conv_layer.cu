#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/conv_layer.hpp"

namespace caffe {


template <typename Dtype>
__global__ void reduce_nmckk_kernel(const int N, const int M, const int C, const int K, const Dtype * a, Dtype * y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N*C) {
    int n = i / C;
    int c = i % C;
    Dtype accum = (Dtype) 0;
    for (int m = 0; m < M; m++) {
      for (int k = 0; k < K; k++) {
        accum += a[(n*M*C*K) + (m*C*K) + (c*K) + k];
      }
    }
    y[i] = accum;
  }
}
template <typename Dtype>
void reduce_nmckk_gpu(const int N, const int M, const int C, const int K, const Dtype * a, Dtype * y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  reduce_nmckk_kernel<Dtype><<<CAFFE_GET_BLOCKS(N*C), CAFFE_CUDA_NUM_THREADS>>>(
      N, M, C, K, a, y);
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const Dtype* bias;
  LayerParameter layer_param(this->layer_param_);
  if (layer_param.phase() == caffe::TRAIN) {
    this->quantize_clock_ += 1;
    this->activation_quantize_clock_ += 1;
  }
  if (this->quantize_term_) {
    Dtype* weight_masked = this->weights_masked_.mutable_gpu_data();
    caffe_gpu_and(this->blobs_[0]->count(), this->quantization_mask, weight, weight_masked);

    if (this->quantize_clock_ == this->quantize_interval_) {
      LOG(INFO) << "Quantizing weights";
      Dtype* weight_mut = this->blobs_[0]->mutable_gpu_data();
      caffe_copy(this->blobs_[0]->count(), weight_masked, weight_mut);
    }
    weight = this->weights_masked_.gpu_data();
  }
  if (this->mask_term_) {
    const Dtype* mask = this->blobs_[this->mask_pos_]->gpu_data();
    Dtype* weight_masked = this->weights_masked_.mutable_gpu_data();
    caffe_gpu_mul(this->blobs_[0]->count(), mask, weight, weight_masked);
    weight = this->weights_masked_.gpu_data();
  }
  if (this->bias_term_) {
    bias = this->blobs_[1]->gpu_data();
    if (this->quantize_term_) {
      Dtype* bias_masked = this->bias_masked_.mutable_gpu_data();
      caffe_gpu_and(this->blobs_[1]->count(), this->quantization_mask, bias, bias_masked);
      if (this->quantize_clock_ == this->quantize_interval_) {
        LOG(INFO) << "Quantizing biases";
        Dtype* bias_mut = this->blobs_[1]->mutable_gpu_data();
        caffe_copy(this->blobs_[1]->count(), bias_masked, bias_mut);
      }
      bias = this->bias_masked_.gpu_data();
    }
    if (this->mask_term_) {
      const Dtype* bias_mask = this->blobs_[this->mask_pos_+1]->gpu_data();
      Dtype* bias_masked = this->bias_masked_.mutable_gpu_data();
      caffe_gpu_mul(this->blobs_[1]->count(), bias_mask, bias, bias_masked);
      bias = this->bias_masked_.gpu_data();
    }
  }
  for (int i = 0; i < bottom.size(); ++i) {
    Dtype* bottom_data = bottom[i]->mutable_gpu_data();
    if (this->activation_quantize_term_ && (this->activation_quantize_clock_ == this->activation_quantize_interval_)) {
      caffe_gpu_and(bottom[i]->count(), this->activation_quantization_mask, bottom_data, bottom_data);
    }
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
  }
  if ((layer_param.phase() == caffe::TRAIN) && (this->quantize_clock_ >= this->quantize_interval_)) {
    this->quantize_clock_ = 0;
  }
  if ((layer_param.phase() == caffe::TRAIN) && (this->activation_quantize_clock_ >= this->activation_quantize_interval_)) {
    this->activation_quantize_clock_ = 0;
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  Dtype* weights_sqr = this->weights_sqr_.mutable_gpu_data();
  Blob<Dtype>  weights_n_masked_;
  Blob<Dtype> bias_n_masked_;
  Blob<Dtype> input_shaped_blob_;
  Dtype* full_weights_diff;

  Dtype* weight_ddiff;
  Dtype* full_weights_ddiff;

  Dtype* bias_diff;
  Dtype* full_bias_diff;
  Dtype* bias_ddiff;
  Dtype* full_bias_ddiff;

  ConvolutionSaliencyParameter conv_saliency_param = this->layer_param_.convolution_saliency_param();

  if (this->saliency_term_ && this->separate_weight_diff_) {
    weights_n_masked_.Reshape({this->num_, this->blobs_[0]->shape()[0], this->blobs_[0]->shape()[1], this->blobs_[0]->shape()[2], this->blobs_[0]->shape()[3]});
    full_weights_diff = weights_n_masked_.mutable_gpu_diff();
  }

  if (this->mask_term_) {
    weight = this->weights_masked_.gpu_data();
  }

  if (this->saliency_term_) {
    if (this->layer_param_.compute_2nd_derivative()) {
      weight_ddiff = this->blobs_[0]->mutable_gpu_diff();
      if (this->separate_weight_diff_) {
        full_weights_ddiff = weights_n_masked_.mutable_gpu_ddiff();
      }
    }
  }

  if (this->bias_term_) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();

    if (this->saliency_term_) {
      if (this->separate_weight_diff_) {
        bias_n_masked_.Reshape({this->num_, this->blobs_[1]->shape()[0]});
        full_bias_diff = bias_n_masked_.mutable_gpu_diff();
      }
    }

    if (this->layer_param_.compute_2nd_derivative()) {
      bias_ddiff = this->blobs_[1]->mutable_gpu_ddiff();
      if (this->separate_weight_diff_) {
        full_bias_ddiff = bias_n_masked_.mutable_gpu_ddiff();
      }
    }
  }

  caffe_gpu_powx(this->blobs_[0]->count(), weight, (Dtype)2, weights_sqr);

  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    const Dtype* top_data = top[i]->gpu_data();
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
    const Dtype* top_ddiff;
    Dtype* bottom_ddiff;
    Dtype* input_sqr_;
    if (this->layer_param_.compute_2nd_derivative()) {
      input_shaped_blob_.Reshape(bottom[i]->shape());
      top_ddiff = top[i]->gpu_ddiff();
      bottom_ddiff = bottom[i]->mutable_gpu_ddiff();
      weight_ddiff = this->blobs_[0]->mutable_gpu_ddiff();
      if (this->separate_weight_diff_) {
        full_weights_ddiff = weights_n_masked_.mutable_gpu_ddiff();
      }
      input_sqr_ = input_shaped_blob_.mutable_gpu_data();
      caffe_gpu_powx(bottom[i]->count(), bottom[i]->gpu_data(), (Dtype) 2, input_sqr_);
    }
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      for (int n = 0; n < this->num_; ++n) {
        if (this->separate_weight_diff_) {
          this->backward_gpu_bias_no_accum(full_bias_diff + n * this->blobs_[1]->count(), top_diff + n * this->top_dim_);
          caffe_gpu_add(this->blobs_[1]->count(), full_bias_diff + n * this->blobs_[1]->count(), bias_diff, bias_diff);
          if (this->layer_param_.compute_2nd_derivative()) {
            this->backward_gpu_bias_no_accum(full_bias_ddiff + n * this->blobs_[1]->count(), top_ddiff + n * this->top_dim_);
            caffe_gpu_add(this->blobs_[1]->count(), full_bias_ddiff + n * this->blobs_[1]->count(), bias_ddiff, bias_ddiff);
          }
        }
        else {
          this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
          if (this->layer_param_.compute_2nd_derivative()) {
            this->backward_gpu_bias(bias_ddiff, top_ddiff + n * this->top_dim_);
          }
        }
      }
      if (this->mask_term_) {
        caffe_gpu_mul(this->blobs_[1]->count(), this->blobs_[this->mask_pos_+1]->gpu_data(), bias_diff, bias_diff);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          if (this->saliency_term_ && this->separate_weight_diff_) {
            this->weight_gpu_gemm_no_accum(bottom_data + n * this->bottom_dim_,
                top_diff + n * this->top_dim_, full_weights_diff + n * this->blobs_[0]->count());
            caffe_gpu_add(this->blobs_[0]->count(), full_weights_diff + n * this->blobs_[0]->count(), weight_diff, weight_diff);
            if (this->layer_param_.compute_2nd_derivative()) {
              this->weight_gpu_gemm_no_accum(input_sqr_ + n * this->bottom_dim_,
                  top_ddiff + n * this->top_dim_, full_weights_ddiff + n * this->blobs_[0]->count());
              caffe_gpu_add(this->blobs_[0]->count(), full_weights_ddiff + n * this->blobs_[0]->count(), weight_ddiff, weight_ddiff);
            }
          }
          else {
            this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
                top_diff + n * this->top_dim_, weight_diff);
            if (this->layer_param_.compute_2nd_derivative()) {
              this->weight_gpu_gemm(input_sqr_ + n * this->bottom_dim_,
                top_ddiff + n * this->top_dim_, weight_ddiff);
            }
          }
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
        if (this->layer_param_.compute_2nd_derivative()) {
          if (propagate_down[i]) {
            this->backward_gpu_gemm(top_ddiff + n * this->top_dim_, weights_sqr,
                bottom_ddiff + n * this->bottom_dim_);
          }
        }
      }
      if (this->mask_term_) {
        // Don't update weights that are masked off
        caffe_gpu_mul(this->blobs_[0]->count(), this->blobs_[this->mask_pos_]->gpu_data(), weight_diff, weight_diff);
      }
    }

    // Compute Channel saliency
    // MULTIPLE INPUTS NOT TREATED
    if (this->saliency_term_) {
      int pos_output_channel_saliency = this->saliency_pos_;
      int pos_input_channel_saliency = this->saliency_pos_;
      if (this->output_channel_saliency_compute_ && this->input_channel_saliency_compute_){
        pos_input_channel_saliency++;
      }
      Dtype * output_channel_saliency_data = NULL;
      Dtype * output_channel_saliency_accum_data = NULL;
      Dtype * input_channel_saliency_data = NULL;
      Dtype * input_channel_saliency_accum_data = NULL;
      Dtype * out_channel_saliency = NULL;
      Dtype * in_channel_saliency = NULL;
      if (this->output_channel_saliency_compute_){
        output_channel_saliency_data = output_saliencies_channel_.mutable_gpu_data();
        output_channel_saliency_accum_data = this->blobs_[pos_output_channel_saliency]->mutable_gpu_data();
      }
      if (this->input_channel_saliency_compute_){
        input_channel_saliency_data = input_saliencies_channel_.mutable_gpu_data();
        input_channel_saliency_accum_data = this->blobs_[pos_input_channel_saliency]->mutable_gpu_data();
      }
      for (int i_s = 0; i_s < conv_saliency_param.saliency_size(); i_s++) {
        if (this->output_channel_saliency_compute_){
          out_channel_saliency = output_channel_saliency_data + (i_s * this->num_output_);
        }
        if (this->input_channel_saliency_compute_){
          in_channel_saliency = input_channel_saliency_data + (i_s * (this->channels_ / this->group_));
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::TAYLOR) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::ACTIVATION)){
            compute_taylor_gpu(bottom_data, bottom_diff, top_data, top_diff, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::HESSIAN_DIAG_APPROX1) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::ACTIVATION)){
            compute_hessian_diag_gpu(bottom_data, bottom_ddiff, top_data, top_ddiff, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::HESSIAN_DIAG_APPROX2) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::ACTIVATION)){
            compute_hessian_diag_approx2_gpu(bottom_data, bottom_diff, top_data, top_diff, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::TAYLOR_2ND_APPROX1) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::ACTIVATION)){
            compute_taylor_2nd_gpu(bottom_data, bottom_diff, bottom_ddiff, top_data, top_diff, top_ddiff, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::TAYLOR_2ND_APPROX2) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::ACTIVATION)){
            compute_taylor_2nd_approx2_gpu(bottom_data, bottom_diff, top_data, top_diff, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::AVERAGE_INPUT) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::ACTIVATION)){
            compute_weight_avg_gpu(bottom_data, top_data, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::APOZ) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::ACTIVATION)){
            compute_apoz_gpu(bottom_data, top_data, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::AVERAGE_GRADIENT) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::ACTIVATION)){
            compute_diff_avg_gpu(bottom_diff, top_diff, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::TAYLOR) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::WEIGHT)){
          compute_taylor_weights_gpu(&weights_n_masked_, &bias_n_masked_, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::HESSIAN_DIAG_APPROX1) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::WEIGHT)){
          compute_hessian_diag_weights_gpu(&weights_n_masked_, &bias_n_masked_, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::HESSIAN_DIAG_APPROX2) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::WEIGHT)){
          compute_hessian_diag_approx2_weights_gpu(&weights_n_masked_, &bias_n_masked_, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::TAYLOR_2ND_APPROX1) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::WEIGHT)){
          compute_taylor_2nd_weights_gpu(&weights_n_masked_, &bias_n_masked_, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::TAYLOR_2ND_APPROX2) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::WEIGHT)){
          compute_taylor_2nd_approx2_weights_gpu(&weights_n_masked_, &bias_n_masked_, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::AVERAGE_INPUT) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::WEIGHT)){
          compute_weight_avg_weights_gpu(&weights_n_masked_, &bias_n_masked_, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::APOZ) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::WEIGHT)){
          compute_apoz_weights_gpu(&weights_n_masked_, &bias_n_masked_, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
        if ((conv_saliency_param.saliency(i_s) == caffe::ConvolutionSaliencyParameter::AVERAGE_GRADIENT) && (conv_saliency_param.saliency_input(i_s) == caffe::ConvolutionSaliencyParameter::WEIGHT)){
          compute_diff_avg_weights_gpu(&weights_n_masked_, &bias_n_masked_, conv_saliency_param.saliency_norm(i_s), in_channel_saliency, out_channel_saliency);
        }
      }
      if (this->layer_param_.convolution_saliency_param().accum()) {
        if (this->output_channel_saliency_compute_) {
          caffe_gpu_add(output_saliencies_channel_.count(), output_channel_saliency_data, output_channel_saliency_accum_data, output_channel_saliency_accum_data);
        }
        if (this->input_channel_saliency_compute_) {
          caffe_gpu_add(input_saliencies_channel_.count(), input_channel_saliency_data, input_channel_saliency_accum_data, input_channel_saliency_accum_data);
        }
      }
      else {
        if (this->output_channel_saliency_compute_) {
          caffe_copy(output_saliencies_channel_.count(), output_channel_saliency_data, output_channel_saliency_accum_data);
        }
        if (this->input_channel_saliency_compute_) {
          caffe_copy(input_saliencies_channel_.count(), input_channel_saliency_data, input_channel_saliency_accum_data);
        }
      }
    }
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::compute_norm_and_batch_avg_gpu(Dtype * in_saliency_data, Dtype * out_saliency_data, caffe::ConvolutionSaliencyParameter::NORM saliency_norm_, Dtype * in_channel_saliency, Dtype * out_channel_saliency) {
  int count, channels;
  Dtype* filter_data, *saliency_data, *channel_saliency;
  if (this->input_channel_saliency_compute_){
    saliency_data = in_saliency_data;
    channel_saliency = in_channel_saliency;
    count = this->input_saliencies_points_.count(2,4);
    filter_data = this->input_saliencies_filter_.mutable_gpu_data();
    channels = this->channels_;
    switch (saliency_norm_) {
      case (caffe::ConvolutionSaliencyParameter::L1): {
        caffe_gpu_abs(this->num_ * channels * count, saliency_data, saliency_data);
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_strided_sum(channels, this->num_, filter_data, filter_data);
    } break;

      case (caffe::ConvolutionSaliencyParameter::L2): {
        caffe_gpu_powx(this->num_ * channels * count, saliency_data, (Dtype) 2, saliency_data);
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_strided_sum(channels, this->num_, filter_data, filter_data);
      } break;

      case (caffe::ConvolutionSaliencyParameter::ABS_SUM): {
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_abs(this->num_ * channels, filter_data, filter_data);
        caffe_gpu_strided_sum(channels, this->num_, filter_data, filter_data);
      } break;

      case (caffe::ConvolutionSaliencyParameter::SQR_SUM): {
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_powx(this->num_ * channels, filter_data, (Dtype) 2, filter_data);
        caffe_gpu_strided_sum(channels, this->num_, filter_data, filter_data);
      } break;

      default: {
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_strided_sum(channels, this->num_, filter_data, filter_data);
      } break;
    }
    caffe_gpu_strided_sum(this->channels_ /this->group_, this->group_, filter_data, channel_saliency);
    caffe_gpu_scal(channels, (Dtype) 1.0 / (Dtype)(this->num_), channel_saliency);
  }
  if (this->output_channel_saliency_compute_){
    saliency_data = out_saliency_data;
    channel_saliency = out_channel_saliency;
    count = this->output_saliencies_points_.count(2,4);
    filter_data = this->output_saliencies_filter_.mutable_gpu_data();
    channels = this->num_output_;
    switch (saliency_norm_) {
      case (caffe::ConvolutionSaliencyParameter::L1): {
        caffe_gpu_abs(this->num_ * channels * count, saliency_data, saliency_data);
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_strided_sum(channels, this->num_, filter_data, channel_saliency);
    } break;

      case (caffe::ConvolutionSaliencyParameter::L2): {
        caffe_gpu_powx(this->num_ * channels * count, saliency_data, (Dtype) 2, saliency_data);
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_strided_sum(channels, this->num_, filter_data, channel_saliency);
      } break;

      case (caffe::ConvolutionSaliencyParameter::ABS_SUM): {
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_abs(this->num_ * channels, filter_data, filter_data);
        caffe_gpu_strided_sum(channels, this->num_, filter_data, channel_saliency);
      } break;

      case (caffe::ConvolutionSaliencyParameter::SQR_SUM): {
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_powx(this->num_ * channels, filter_data, (Dtype) 2, filter_data);
        caffe_gpu_strided_sum(channels, this->num_, filter_data, channel_saliency);
      } break;

      default: {
        caffe_gpu_sum(this->num_ * channels, count, saliency_data, filter_data); //sum hxw
        caffe_gpu_strided_sum(channels, this->num_, filter_data, channel_saliency);
      } break;
    }
    caffe_gpu_scal(channels, (Dtype) 1.0 / (Dtype)(this->num_), channel_saliency);
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::compute_norm_and_batch_avg_weights_gpu(Dtype * weight_saliency_data, Dtype * bias_saliency_data, caffe::ConvolutionSaliencyParameter::NORM saliency_norm_, Dtype * in_channel_saliency, Dtype * out_channel_saliency) {

  Dtype* filter_data;

  int kernel_size = this->blobs_[0]->count(2,4);
  int weights_count = this->blobs_[0]->count();
  int bias_count;

  if (this->bias_term_) {
    bias_count = this->blobs_[1]->count();
  }
  switch (saliency_norm_) {
    case (caffe::ConvolutionSaliencyParameter::L1): {
      caffe_gpu_abs(this->num_ * weights_count, weight_saliency_data, weight_saliency_data);
      if (this->output_channel_saliency_compute_){
        filter_data = output_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_, this->channels_ * kernel_size / this->group_, weight_saliency_data, filter_data);
        if (this->saliency_bias_ && this->bias_term_ && bias_saliency_data != NULL){
          caffe_gpu_abs(this->num_ * bias_count, bias_saliency_data, bias_saliency_data);
          caffe_gpu_add(this->num_ * bias_count, bias_saliency_data, filter_data, filter_data);
        }
        caffe_gpu_strided_sum(this->num_output_, this->num_, filter_data, out_channel_saliency);
        caffe_gpu_scal(this->num_output_, (Dtype) 1.0 / (Dtype)(this->num_), out_channel_saliency);
      }
      if (this->input_channel_saliency_compute_) {
        filter_data = input_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_ * this->channels_ / this->group_, kernel_size, weight_saliency_data, weight_saliency_data);
        caffe_gpu_strided_sum_inner(this->num_, this->num_output_, this->channels_ / this->group_, weight_saliency_data, filter_data);
        caffe_gpu_strided_sum(this->channels_ / this->group_, this->num_, filter_data, in_channel_saliency);
        caffe_gpu_scal(this->channels_ / this->group_, (Dtype) 1.0 / (Dtype)(this->num_), in_channel_saliency);
      }
    } break;

    case (caffe::ConvolutionSaliencyParameter::L2): {
      caffe_gpu_powx(this->num_ * weights_count, weight_saliency_data, (Dtype) 2, weight_saliency_data);
      if (this->output_channel_saliency_compute_){
        filter_data = output_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_, this->channels_ * kernel_size / this->group_, weight_saliency_data, filter_data);
        if (this->saliency_bias_ && this->bias_term_ && bias_saliency_data != NULL){
          caffe_gpu_powx(this->num_ * bias_count, bias_saliency_data, (Dtype) 2, bias_saliency_data);
          caffe_gpu_add(this->num_ * bias_count, bias_saliency_data, filter_data, filter_data);
        }
        caffe_gpu_strided_sum(this->num_output_, this->num_, filter_data, out_channel_saliency);
        caffe_gpu_scal(this->num_output_, (Dtype) 1.0 / (Dtype)(this->num_), out_channel_saliency);
      }
      if (this->input_channel_saliency_compute_) {
        filter_data = input_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_ * this->channels_ / this->group_, kernel_size, weight_saliency_data, weight_saliency_data);
        caffe_gpu_strided_sum_inner(this->num_, this->num_output_, this->channels_ / this->group_, weight_saliency_data, filter_data);
        caffe_gpu_strided_sum(this->channels_ / this->group_, this->num_, filter_data, in_channel_saliency);
        caffe_gpu_scal(this->channels_ / this->group_, (Dtype) 1.0 / (Dtype)(this->num_), in_channel_saliency);
      }
    } break;

    case (caffe::ConvolutionSaliencyParameter::ABS_SUM): {
      if (this->output_channel_saliency_compute_){
        filter_data = output_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_, this->channels_ * kernel_size / this->group_, weight_saliency_data, filter_data);
        if (this->saliency_bias_ && this->bias_term_ && bias_saliency_data != NULL){
          caffe_gpu_add(this->num_ * bias_count, bias_saliency_data, filter_data, filter_data);
        }
        caffe_gpu_abs(this->num_ * this->num_output_, filter_data, filter_data);
        caffe_gpu_strided_sum(this->num_output_, this->num_, filter_data, out_channel_saliency);
        caffe_gpu_scal(this->num_output_, (Dtype) 1.0 / (Dtype)(this->num_), out_channel_saliency);
      }
      if (this->input_channel_saliency_compute_) {
        filter_data = input_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_ * this->channels_ / this->group_, kernel_size, weight_saliency_data, weight_saliency_data);
        caffe_gpu_strided_sum_inner(this->num_, this->num_output_, this->channels_ / this->group_, weight_saliency_data, filter_data);
        caffe_gpu_abs(this->num_ * this->channels_ / this->group_, filter_data, filter_data);
        caffe_gpu_strided_sum(this->channels_ / this->group_, this->num_, filter_data, in_channel_saliency);
        caffe_gpu_scal(this->channels_ / this->group_, (Dtype) 1.0 / (Dtype)(this->num_), in_channel_saliency);
      }
    } break;

    case (caffe::ConvolutionSaliencyParameter::SQR_SUM): {
      if (this->output_channel_saliency_compute_){
        filter_data = output_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_, this->channels_ * kernel_size / this->group_, weight_saliency_data, filter_data);
        if (this->saliency_bias_ && this->bias_term_ && bias_saliency_data != NULL){
          caffe_add(this->num_ * bias_count, bias_saliency_data, filter_data, filter_data);
        }
        caffe_gpu_powx(this->num_ * this->num_output_, filter_data, (Dtype) 2, filter_data);
        caffe_gpu_strided_sum(this->num_output_, this->num_, filter_data, out_channel_saliency);
        caffe_gpu_scal(this->num_output_, (Dtype) 1.0 / (Dtype)(this->num_), out_channel_saliency);
      }
      if (this->input_channel_saliency_compute_) {
        filter_data = input_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_ * this->channels_ / this->group_, kernel_size, weight_saliency_data, weight_saliency_data);
        caffe_gpu_strided_sum_inner(this->num_, this->num_output_, this->channels_ / this->group_, weight_saliency_data, filter_data);
        caffe_gpu_powx(this->num_ * this->channels_ / this->group_, filter_data, (Dtype) 2, filter_data);
        caffe_gpu_strided_sum(this->channels_ / this->group_, this->num_, filter_data, in_channel_saliency);
        caffe_gpu_scal(this->channels_ / this->group_, (Dtype) 1.0 / (Dtype)(this->num_), in_channel_saliency);
      }
    } break;

    default: {
      if (this->output_channel_saliency_compute_){
        filter_data = output_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_, this->channels_ * kernel_size / this->group_, weight_saliency_data, filter_data);
        if (this->saliency_bias_ && this->bias_term_ && bias_saliency_data != NULL){
          caffe_gpu_add(this->num_ * bias_count, bias_saliency_data, filter_data, filter_data);
        }
        caffe_gpu_strided_sum(this->num_output_, this->num_, filter_data, out_channel_saliency);
        caffe_gpu_scal(this->num_output_, (Dtype) 1.0 / (Dtype)(this->num_), out_channel_saliency);
      }
      if (this->input_channel_saliency_compute_) {
        filter_data = input_saliencies_filter_.mutable_gpu_data();
        caffe_gpu_sum(this->num_ * this->num_output_ * this->channels_ / this->group_, kernel_size, weight_saliency_data, weight_saliency_data);
        caffe_gpu_strided_sum_inner(this->num_, this->num_output_, this->channels_ / this->group_, weight_saliency_data, filter_data);
        caffe_gpu_strided_sum(this->channels_ / this->group_, this->num_, filter_data, in_channel_saliency);
        caffe_gpu_scal(this->channels_ / this->group_, (Dtype) 1.0 / (Dtype)(this->num_), in_channel_saliency);
      }
    } break;
  }
}

template void ConvolutionLayer<float>::compute_norm_and_batch_avg_gpu(float * in_saliency_data, float * out_saliency_data, caffe::ConvolutionSaliencyParameter::NORM saliency_norm_, float * in_channel_saliency, float * out_channel_saliency);
template void ConvolutionLayer<double>::compute_norm_and_batch_avg_gpu(double * in_saliency_data, double * out_saliency_data, caffe::ConvolutionSaliencyParameter::NORM saliency_norm_, double * in_channel_saliency, double * out_channel_saliency);
template void ConvolutionLayer<float>::compute_norm_and_batch_avg_weights_gpu(float * weight_saliency_data, float * bias_saliency_data, caffe::ConvolutionSaliencyParameter::NORM saliency_norm_, float * in_channel_saliency, float * out_channel_saliency);
template void ConvolutionLayer<double>::compute_norm_and_batch_avg_weights_gpu(double * weight_saliency_data, double * bias_saliency_data, caffe::ConvolutionSaliencyParameter::NORM saliency_norm_, double * in_channel_saliency, double * out_channel_saliency);

#ifdef CPU_ONLY
STUB_GPU(ConvolutionLayer);
#endif


INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
