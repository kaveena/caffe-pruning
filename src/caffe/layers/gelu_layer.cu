#include "hip/hip_runtime.h"
// GELU neuron activation function layer.

#include <vector>

#include "caffe/layers/gelu_layer.hpp"

// approximation of \sqrt{\frac{2}{\pi}}
#define SQRT_2_over_pi 0.405285

namespace caffe {

template <typename Dtype>
__global__ void GELUForward(const int n, const Dtype* in, Dtype* tanhx_data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    tanhx_data[index] = tanh((SQRT_2_over_pi * in[index]) + (SQRT_2_over_pi * 0.044715 * in[index] * in[index] * in[index]));
    out[index] = 0.5 * in[index] * ( 1 + tanhx_data[index] );
  }
}

template <typename Dtype>
void GELULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* tanhx_data = tanhx.mutable_gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  GELUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, tanhx_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void GELUBackward(const int n, const Dtype* bottom_data,
    const Dtype* top_diff, const Dtype* tanhx_data, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype x = bottom_data[index];
    bottom_diff[index] = top_diff[index] * (1 - (tanhx_data[index] * tanhx_data[index]));
    bottom_diff[index] = 0.5 + (0.5 * tanhx_data[index]) + (0.5 * SQRT_2_over_pi * bottom_diff[index] * (x + 0.134145 * x * x * x));
  }
}

template <typename Dtype>
void GELULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* tanhx_data = tanhx.gpu_data();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    GELUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_diff, tanhx_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(GELULayer);

}  // namespace caffe
